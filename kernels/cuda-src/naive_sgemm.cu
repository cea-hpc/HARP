
#include <hip/hip_runtime.h>
#include <cstdint>

extern "C" __global__
auto naive_sgemm(
    uint64_t m, uint64_t n, uint64_t k,
    float alpha,
    float const* A, uint64_t lda,
    float const* B, uint64_t ldb,
    float beta,
    float* C, uint64_t ldc
) -> void {
    uint64_t i = threadIdx.y;
    uint64_t j = threadIdx.x;

    float acc = 0.0;
    for (size_t l = 0; l < k; ++l) {
        acc += A[i * lda + l] * B[l * ldb + j];
    }
    C[i* ldc + j] *= beta + alpha * acc;
}
