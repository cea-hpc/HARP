
#include <hip/hip_runtime.h>
#include <cstdint>

constexpr uint64_t BLOCK_SIZE = 32;

extern "C" __global__
auto tiled_sgemm(
    uint64_t m, uint64_t n, uint64_t k,
    float alpha,
    float const* A, uint64_t lda,
    float const* B, uint64_t ldb,
    float beta,
    float* C, uint64_t ldc
) -> void {
    uint64_t block_row = blockIdx.y;
    uint64_t block_col = blockIdx.x;
    uint64_t row = threadIdx.y;
    uint64_t col = threadIdx.x;

    float* Ct = &C[block_row * ldc * BLOCK_SIZE + block_col * BLOCK_SIZE];

    float acc = 0.0;
    for (size_t t = 0; t < (k / BLOCK_SIZE); ++t) {
        __shared__ float At[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bt[BLOCK_SIZE][BLOCK_SIZE];

        At[row][col] = A[block_row * lda * BLOCK_SIZE + t * BLOCK_SIZE];
        Bt[row][col] = B[t * ldb * BLOCK_SIZE + block_col * BLOCK_SIZE];
        __syncthreads();

        for (size_t l = 0; l < BLOCK_SIZE; ++l) {
            acc += At[row][l] * Bt[l][col];
        }
        __syncthreads();
    }

    Ct[row * ldc + col] *= beta + alpha * acc;
}
